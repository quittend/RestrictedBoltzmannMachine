#include "hip/hip_runtime.h"
#include <sstream> //stringstream
#include <string> //string
#include <iomanip> //setprecision
#include <fstream> //ofstream
#include <cmath> //floor
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include "Utility.cuh"
#include "RestrictedBoltzmannMachine.cuh"


void RestrictedBoltzmannMachine::SampleFromHiddenGivenVisible(double* const hiddenMatrixDevice, const double* const visibleMatrixDevice, const double* resizedHiddenBiasesDevice, const int samples) const {
	static const double alpha = 1.0;
	static const double beta = 0.0;
	hipblasDgemm(defaultHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, samples, numHidden, numVisible, &alpha, visibleMatrixDevice, samples, weightsMatrixDevice, numVisible, &beta, hiddenMatrixDevice, samples);
	hipblasDaxpy(defaultHandle, samples * numHidden, &alpha, resizedHiddenBiasesDevice, 1, hiddenMatrixDevice, 1);

	const auto itDevice = thrust::device_pointer_cast(hiddenMatrixDevice);
	thrust::transform(itDevice, itDevice + samples*numHidden, itDevice, [] __device__ (double x) { return Utility::randFromBinomial(Utility::sigmoidFunction(x)); });
}

void RestrictedBoltzmannMachine::SampleFromVisibleGivenHidden(double* const visibleMatrixDevice, const double* const hiddenMatrixDevice, const double* resizedVisibleBiasesDevice, const int samples) const {
	static const double alpha = 1.0;
	static const double beta = 0.0;
	hipblasDgemm(defaultHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, samples, numVisible, numHidden, &alpha, hiddenMatrixDevice, samples, weightsMatrixDevice, numVisible, &beta, visibleMatrixDevice, samples);
	hipblasDaxpy(defaultHandle, samples * numVisible, &alpha, resizedVisibleBiasesDevice, 1, visibleMatrixDevice, 1);
	
	const auto itDevice = thrust::device_pointer_cast(visibleMatrixDevice);
	thrust::transform(itDevice, itDevice + samples*numVisible, itDevice, [] __device__ (double x) { return Utility::randFromBinomial(Utility::sigmoidFunction(x)); });
}

Utility::Gradients RestrictedBoltzmannMachine::ContrastiveDivergence(double* const dataDevice, const int samples, const int cdSteps) const
{
	// Memory allocation
	double* resizedHiddenBiasesDevice; hipMalloc(&resizedHiddenBiasesDevice, sizeof(double)*samples*numHidden);
	double* resizedVisibleBiasesDevice;  hipMalloc(&resizedVisibleBiasesDevice, sizeof(double)*samples*numVisible);

	double* sampleVisiblePositiveDevice; hipMalloc(&sampleVisiblePositiveDevice, sizeof(double) * samples * numVisible);
	double* sampleHiddenPositiveDevice;	hipMalloc(&sampleHiddenPositiveDevice, sizeof(double) * samples * numHidden);
	double* sampleVisibleNegativeDevice; hipMalloc(&sampleVisibleNegativeDevice, sizeof(double) * samples * numVisible);
	double* sampleHiddenNegativeDevice;	hipMalloc(&sampleHiddenNegativeDevice, sizeof(double) * samples * numHidden);

	hipStream_t* streams = new hipStream_t[samples];
	hipblasHandle_t* handles = new hipblasHandle_t[samples];
	for (int i = 0; i < samples; i++) {
		hipblasCreate(&handles[i]);
		hipStreamCreate(&streams[i]);
		hipblasSetStream(handles[i], streams[i]);
	}

	for (int i = 0; i < samples; i++) {
		hipblasDcopy(handles[i], numHidden, hiddenBiasesDevice, 1, resizedHiddenBiasesDevice + i, samples);
		hipblasDcopy(handles[i], numVisible, visibleBiasesDevice, 1, resizedVisibleBiasesDevice + i, samples);
		hipblasDcopy(defaultHandle, numVisible, dataDevice + i*numVisible, 1, sampleVisiblePositiveDevice + i, samples);
		hipStreamDestroy(streams[i]);
	}

	// Sampling
	SampleFromHiddenGivenVisible(sampleHiddenPositiveDevice, sampleVisiblePositiveDevice, resizedHiddenBiasesDevice, samples);
	for (int i = 0; i < cdSteps; i++) {
		SampleFromVisibleGivenHidden(sampleVisibleNegativeDevice, i == 0 ? sampleHiddenPositiveDevice : sampleHiddenNegativeDevice, resizedVisibleBiasesDevice, samples);
		SampleFromHiddenGivenVisible(sampleHiddenNegativeDevice, sampleVisibleNegativeDevice, resizedHiddenBiasesDevice, samples);
	}

	// Memory deallocation
	hipFree(resizedHiddenBiasesDevice);
	hipFree(resizedVisibleBiasesDevice);
	delete[] handles;
	delete[] streams;

	// Results
	return Utility::Gradients(sampleVisiblePositiveDevice, sampleHiddenPositiveDevice, sampleVisibleNegativeDevice, sampleHiddenNegativeDevice);
}

RestrictedBoltzmannMachine::RestrictedBoltzmannMachine(const int hidden, const int visible) {
	numVisible = visible;
	numHidden = hidden;

	// Allocate memory
	weightsMatrix = new double[hidden * visible]();
	visibleBiases = new double[visible]();
	hiddenBiases = new double[hidden]();
	hipMalloc(&weightsMatrixDevice, sizeof(double)*visible*hidden);
	hipMalloc(&hiddenBiasesDevice, sizeof(double)*hidden);
	hipMalloc(&visibleBiasesDevice, sizeof(double)*visible);

	//For test run
	//for (int i = 0; i < hidden*visible; i++)
	//	weightsMatrix[i] = i+1;
	//std::fill_n(visibleBiases, visible, 1);
	//std::fill_n(hiddenBiases, hidden, 1);

	//For final run
	Utility::randVector(weightsMatrix, hidden * visible);
	Utility::randVector(visibleBiases, visible);
	Utility::randVector(hiddenBiases, hidden);

	// Copy data to a device
	hipMemcpy(weightsMatrixDevice, weightsMatrix, sizeof(double)*visible*hidden, hipMemcpyHostToDevice);
	hipMemcpy(visibleBiasesDevice, visibleBiases, sizeof(double)*visible, hipMemcpyHostToDevice);
	hipMemcpy(hiddenBiasesDevice, hiddenBiases, sizeof(double)*hidden, hipMemcpyHostToDevice);

	// Handle creation
	hipblasCreate(&defaultHandle);
}

RestrictedBoltzmannMachine::~RestrictedBoltzmannMachine() {
	delete[] weightsMatrix;
	delete[] visibleBiases;
	delete[] hiddenBiases;
	hipFree(weightsMatrixDevice);
	hipFree(visibleBiasesDevice);
	hipFree(hiddenBiasesDevice);
	hipblasDestroy(defaultHandle);
}


Utility::Deltas RestrictedBoltzmannMachine::GetDeltaWeights(Utility::Gradients& gradients, const int samples, const double learningRate) const {
	// Memory allocation
	double* deltaWeightsMatrix1Device; hipMalloc(&deltaWeightsMatrix1Device, sizeof(double)*numHidden*numVisible);
	double* deltaWeightsMatrix2Device; hipMalloc(&deltaWeightsMatrix2Device, sizeof(double)*numHidden*numVisible);
	double* deltaVisibleBiasesDevice; hipMalloc(&deltaVisibleBiasesDevice, sizeof(double)*numVisible);
	double* deltaHiddenBiasesDevice; hipMalloc(&deltaHiddenBiasesDevice, sizeof(double)*numHidden);
	thrust::device_ptr<double> it = thrust::device_pointer_cast(deltaWeightsMatrix1Device); 
	thrust::fill_n(it, numHidden*numVisible, 0.0);
	it = thrust::device_pointer_cast(deltaWeightsMatrix2Device); thrust::fill_n(it, numHidden*numVisible, 0.0);
	it = thrust::device_pointer_cast(deltaVisibleBiasesDevice); thrust::fill_n(it, numVisible, 0.0);
	it = thrust::device_pointer_cast(deltaHiddenBiasesDevice); thrust::fill_n(it, numHidden, 0.0);

	hipStream_t* streams = new hipStream_t[samples];
	hipblasHandle_t* handles = new hipblasHandle_t[samples];
	for (int i = 0; i < samples; i++) {
		hipblasCreate(&handles[i]);
		hipStreamCreate(&streams[i]);
		hipblasSetStream(handles[i], streams[i]);
	}

	// Weights (might be and issue, because Kosinski said so)
	const double alpha = learningRate / samples;
	for (int i = 0; i < samples; i++) {
		double* v = gradients.visiblePositiveGradientsDevice + i;
		double* h = gradients.hiddenPositiveGradientsDevice + i;
		double* vprim = gradients.visibleNegativeGradientsDevice + i;
		double* hprim = gradients.hiddenNegativeGradientsDevice + i;

		hipblasDger(handles[i], numVisible, numHidden, &alpha, v, samples, h, samples, deltaWeightsMatrix1Device, numVisible);
		hipblasDger(handles[i], numVisible, numHidden, &alpha, vprim, samples, hprim, samples, deltaWeightsMatrix2Device, numVisible);
		hipStreamDestroy(streams[i]);
	}

	delete[] handles;
	delete[] streams;

	static const double minusOne = -1.0;
	hipblasDaxpy(defaultHandle, numHidden * numVisible, &minusOne, deltaWeightsMatrix2Device, 1, deltaWeightsMatrix1Device, 1);
	hipFree(deltaWeightsMatrix2Device);

	// Additional memory allocation for biases
	double* onesDevice; hipMalloc(&onesDevice, sizeof(double)*samples);
	it = thrust::device_pointer_cast(onesDevice); thrust::fill_n(it, samples, 1.0);

	hipStream_t streamsBiases[2];
	hipblasHandle_t handlesBiases[2];

	for (int i = 0; i < 2; i++) {
		hipblasCreate(&handlesBiases[i]);
		hipStreamCreate(&streamsBiases[i]);
		hipblasSetStream(handlesBiases[i], streamsBiases[i]);
	}

	static const double beta = 0.0;
	// Visible biases
	hipblasDaxpy(handlesBiases[0], samples * numVisible, &minusOne, gradients.visibleNegativeGradientsDevice, 1, gradients.visiblePositiveGradientsDevice, 1);
	hipblasDgemv(handlesBiases[0], HIPBLAS_OP_T, samples, numVisible, &alpha, gradients.visiblePositiveGradientsDevice, samples, onesDevice, 1, &beta, deltaVisibleBiasesDevice, 1);
	
	// Hidden biases
	hipblasDaxpy(handlesBiases[1], samples * numHidden, &minusOne, gradients.hiddenNegativeGradientsDevice, 1, gradients.hiddenPositiveGradientsDevice, 1);
	hipblasDgemv(handlesBiases[1], HIPBLAS_OP_T, samples, numHidden, &alpha, gradients.hiddenPositiveGradientsDevice, samples, onesDevice, 1, &beta, deltaHiddenBiasesDevice, 1);

	hipStreamDestroy(streamsBiases[0]);
	hipStreamDestroy(streamsBiases[1]);

	// Additional memory deallocation for biases
	hipFree(onesDevice);

	// Results
	return Utility::Deltas(deltaWeightsMatrix1Device, deltaVisibleBiasesDevice, deltaHiddenBiasesDevice);
}

Utility::Deltas RestrictedBoltzmannMachine::TrainingIteration(double* const dataDevice, const int samples, const double learningRate, const int cdSteps) const {
	Utility::Gradients g = ContrastiveDivergence(dataDevice, samples, cdSteps);
	return GetDeltaWeights(g, samples, learningRate);
}

void RestrictedBoltzmannMachine::TrainNetwork(double* const dataDevice, const int samples, const double learningRate, const int miniBatchSize, const int epochs, const int cdSteps) {
	int numMiniBatches = static_cast<int>(ceil(samples * 1.0 / miniBatchSize));
	for (int i = 0; i < epochs; i++) {
		for (int j = 0; j < numMiniBatches; j++) {
			int currTranslation = j*miniBatchSize;
			int nextTranslation = (j + 1)*miniBatchSize;
			int currMiniBatch = nextTranslation > samples ? samples - currTranslation : miniBatchSize;

			Utility::Deltas d = TrainingIteration(dataDevice + currTranslation*numVisible, currMiniBatch, learningRate, cdSteps);

			static const double alpha = 1.0;
			hipblasDaxpy(defaultHandle, numHidden * numVisible, &alpha, d.deltaWeightsMatrixDevice, 1, weightsMatrixDevice, 1);
			hipblasDaxpy(defaultHandle, numHidden, &alpha, d.deltaHiddenBiasesDevice, 1, hiddenBiasesDevice, 1);
			hipblasDaxpy(defaultHandle, numVisible, &alpha, d.deltaVisibleBiasesDevice, 1, visibleBiasesDevice, 1);
		}
	}

	hipMemcpy(weightsMatrix, weightsMatrixDevice, sizeof(double)*numVisible*numHidden, hipMemcpyDeviceToHost);
	hipMemcpy(hiddenBiases, hiddenBiasesDevice, sizeof(double)*numHidden, hipMemcpyDeviceToHost);
	hipMemcpy(visibleBiases, visibleBiasesDevice, sizeof(double)*numVisible, hipMemcpyDeviceToHost);
}

std::string RestrictedBoltzmannMachine::GetWeights() const {
	std::stringstream ss;
	for (int i = 0; i < numVisible; i++) {
		for (int j = 0; j < numHidden; j++)
			ss << std::setprecision(2) << std::fixed << (weightsMatrix[j + numHidden*i] >= 0.0 ? "+" : "") << weightsMatrix[j + numHidden*i] << " ";
		ss << std::endl;
	}

	return ss.str();
}

void RestrictedBoltzmannMachine::SaveWeights(const std::string filename) const {
	std::ofstream out(filename);
	out << GetWeights();
	out.close();
}